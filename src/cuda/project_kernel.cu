#include "hip/hip_runtime.h"
//#include <hip/hip_runtime_api.h>
#include "project_kernel.cuh"
#include <stdio.h>

__constant__ float K[3][3];
__constant__ float Kinv[3][3];
__constant__ float eps2;
__constant__ int npoints;
__device__  float image[480*640];

__global__ void project_kernel(float *d_x, float *d_y, float *d_z, float *d_nx,
                               float *d_ny, float *d_nz, float *d_r, float *d_rgba)
{
    // each kernel function gets a surfel in the cloud

    // assume global K, assume points have already been transformed into camera frame?

    // now, based on normal and radius determine the pixels covered
    // we could probably use the projection matrix on the ellipse somehow
    //int tid = threadIdx.x + blockDim.x*blockIdx.x;
    int i = threadIdx.x + blockDim.x*blockIdx.x;

    int index = 30000;

    if (i == index) {
        printf("npoints: %d\n", npoints);
        printf("eps2: %f\n", eps2);
    }

    if (i >= npoints || d_z[i] < 0.0f) {
        if (i == index) {
            printf("Index: %d, Depth: %f\n", i, d_z[i]);
            printf("K: [%f, %f, %f; %f, %f, %f; %f, %f, %f]\n", K[0][0], K[0][1], K[0][2], K[1][0], K[1][1], K[1][2], K[2][0], K[2][1], K[2][2]);
        }
        return;
    }

    float px = K[0][0]*d_x[i]/d_z[i] + K[0][2];
    float py = K[1][1]*d_y[i]/d_z[i] + K[1][2];
    if (px < 0 || px > 640 || py < 0 || py > 480) {
        return;
    }

    float r2 = 1.0f/(d_r[i]*d_r[i]);

    float NN[3][3];
    NN[0][0] = d_nx[i]*d_nx[i]; NN[0][1] = d_nx[i]*d_ny[i]; NN[0][2] = d_nx[i]*d_nz[i];
    NN[1][0] = d_ny[i]*d_nx[i]; NN[1][1] = d_ny[i]*d_ny[i]; NN[1][2] = d_ny[i]*d_nz[i];
    NN[2][0] = d_nz[i]*d_nx[i]; NN[2][1] = d_nz[i]*d_ny[i]; NN[2][2] = d_nz[i]*d_nz[i];

    float A[3][3];
    for (int row = 0; row < 3; ++row) {
        for (int col = 0; col < 3; ++col) {
            A[row][col] = (eps2 - r2)*NN[row][col];
        }
        A[row][row] += r2;
    }

    float c[3] = {d_x[i], d_y[i], d_z[i]};

    float Ac[3];
    for (int row = 0; row < 3; ++row) {
        Ac[row] = 0.0f;
        for (int col = 0; col < 3; ++col) {
            Ac[row] += A[row][col]*c[col];
        }
    }

    float cAc = 0.0f;
    for (int row = 0; row < 3; ++row) {
        cAc += c[row]*Ac[row];
    }

    // now we reuse A to compute AA
    for (int row = 0; row < 3; ++row) {
        for (int col = 0; col < 3; ++col) {
            A[row][col] = (cAc - 1.0f)*A[row][col] - Ac[row]*Ac[col];
        }
    }

    // and finally, let's reuse NN and A to compute the AA with camera matrix
    for (int row = 0; row < 3; ++row) {
        for (int col = 0; col < 3; ++col) {
            // A*Kinv
            NN[row][col] = A[row][0]*Kinv[0][col] + A[row][1]*Kinv[1][col] + A[row][2]*Kinv[2][col];
        }
    }

    for (int row = 0; row < 3; ++row) {
        for (int col = 0; col < 3; ++col) {
            // this time Kinv^T*A
            A[row][col] = Kinv[0][row]*NN[0][col] + Kinv[1][row]*NN[1][col] + Kinv[2][row]*NN[2][col];
        }
    }

    int minx = 640; int maxx = 0; int miny = 480; int maxy = 0;
    float ox[4] = {d_r[i], -d_r[i], 0.0f, 0.0f};
    float oy[4] = {0.0f, 0.0f, d_r[i], -d_r[i]};

    for (int col = 0; col < 4; ++col) {
        px = K[0][0]*(d_x[i]+ox[col])/d_z[i] + K[0][2];
        py = K[1][1]*(d_y[i]+oy[col])/d_z[i] + K[1][2];
        if (px < minx) {
            minx = px;
        }
        if (py < miny) {
            miny = py;
        }
        if (px > maxx) {
            maxx = px;
        }
        if (py > maxy) {
            maxy = py;
        }
    }

    float x, y, disc;
    for (int row = miny; row < maxy; ++row) {

        for (int col = minx; col < maxx; ++col) {
            x = col; y = row;
            // C2 + B2.transpose()*v + v.transpose()*A2*v
            disc = A[2][2] + 2.0f*(A[0][2]*x + A[1][2]*y) + A[0][0]*x*x + 2.0f*A[0][1]*x*y + A[1][1]*y*y;
            if (disc < 0.0f) {

                if (i == index) {
                    printf("disc: %f!", disc);
                }

                x = 1.0f;
                image[640*row+col] = d_rgba[i];
            }
        }
    }

}

void project(float *d_x, float *d_y, float *d_z, float *d_nx,
             float *d_ny, float *d_nz, float *d_r, float *d_rgba,
             int nblocks, int nthreads, float neps2, float* nK, float* nKinv, float* himage,
             int hnpoints)
{
    hipMemcpyToSymbol(HIP_SYMBOL(K), nK, 9*sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(Kinv), nKinv, 9*sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(eps2), &neps2, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(npoints), &hnpoints, sizeof(int));
    project_kernel<<<nblocks,nthreads>>>(d_x, d_y, d_z, d_nx, d_ny, d_nz, d_r, d_rgba);
    //float *my_image;
    //hipGetSymbolAddress((void **)&my_image, image);
    hipMemcpyFromSymbol(himage, HIP_SYMBOL(image), 480*640*sizeof(float));
}
